#include <cstdlib>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <vector>


//constants for matrix dimensions and tile size
const static int width = 4096; //Width of the matrix
const static int height = 4096; //Height of the matrix
const static int tile_dim = 16; //Tile dimension of the shared memory


//Optimized transpose kernel using shared memory
__global__ void transpose_SM_kernel(float *in, float *out, int width, int height){
    //Declare a 2D shared memory tile of size tile_dim * tile_dim
    __shared__ float tile[tile_dim][tile_dim];

    //calculate the indices for the input matrix
    int x_tile_index = blockIdx.x * tile_dim; //start column of the tile in the input
    int y_tile_index = blockIdx.y * tile_dim; //Start row of the tile in the input


    //compute global input index for the current thread
    int in_index = (y_tile_index + threadIdx.y) * width + (x_tile_index + threadIdx.x);

    //Load the data from the global memory into shared memory
    tile[threadIdx.y][threadIdx.x] = in[in_index];

    //Synchronize threads to ensure all data is loaded into shared memory
    __syncthreads();

    //Calculate the indices for the output matrix (transpose the tile)
    int x_transpose_tile_index = blockIdx.y * tile_dim; // Transposed start column
    int y_transpose_tile_index = blockIdx.x * tile_dim; // transpose start row

    //compute global output index for the current thread
    int out_index = (x_transpose_tile_index + threadIdx.y) * height + (y_transpose_tile_index + threadIdx.x);

    // Write data from shared memory to global memory (transposed)
    out[out_index] = tile[threadIdx.x][threadIdx.y];

}


int main() {
    //Host machines 
    std::vector<float> matrix_in;
    std::vector<float> matrix_out;

    //Resize matrices to match the dimensions
    matrix_in.resize(width * height);
    matrix_out.resize(width * height);


    //Initialize input matrix with random values
    for (int i = 0; i < width * height; i++){
        matrix_in[i] = (float)rand() / (float)RAND_MAX;
    }

    //Device pointers
    float *d_in, *d_out;

    //Allocate memory on the device for input and output matrices
    hipMalloc((void **)&d_in, width * height * sizeof(float));
    hipMalloc((void **)&d_out, width * height * sizeof(float));

    //copy input marix from host to device
    hipMemcpy(d_in, matrix_in.data(), width * height * sizeof(float), hipMemcpyHostToDevice);

    //Kernel configuration 
    int block_x  = width / tile_dim;
    int block_y = height / tile_dim;

    dim3 grid(block_x, block_y); //Grid of blocks
    dim3 block(tile_dim, tile_dim); //Threads per block(tile_dim * tile_dim)

    //Create CUDA eventes for timing
    hipEvent_t start_kernel_event, end_kernel_event;
    hipEventCreate(&start_kernel_event);
    hipEventCreate(&end_kernel_event);

    //Warm up GPU 
    printf("Warming up GPU with kernel execution...\n");
    for ( int i = 0; i < 10 ; i++){
        transpose_SM_kernel<<<grid, block >>>(d_in, d_out, width, height);
    }

    //Record start time
    hipEventRecord(start_kernel_event, 0);

    //Launch the transpose kernel multiple times for averaging
    for( int i = 0; i < 10; i++){
        transpose_SM_kernel<<<grid, block>>>(d_in, d_out, width, height);
    }

    //Record end time and synchronize
    hipEventRecord(end_kernel_event, 0);
    hipEventSynchronize(end_kernel_event);

    //Calculate elapsed time
    float time_kernel;
    hipEventElapsedTime(&time_kernel, start_kernel_event, end_kernel_event);

    //Priting timing results
    printf("Kernel Execution Complete. \n");
    printf("Execution time (average over 10 runs): %.6f ms\n", time_kernel / 10);
    printf("Bandwidth: %.6f GB/s\n",
           2.0 * 10000 * (((double)(width) * (double)height) * sizeof(float)) /
               (time_kernel * 1024 * 1024 * 1024));

    //copy transposed matrix from device to host
    hipMemcpy(matrix_out.data(), d_out, width * height * sizeof(float), hipMemcpyDeviceToHost);

    //Free device memory 
    hipFree(d_in);
    hipFree(d_out);

    return 0;

}