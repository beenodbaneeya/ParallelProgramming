#include <cstdlib>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <vector>

//constants for matrix dimensions and tile sizes
const static int width = 4096;  //width of matrix
const static int height = 4096; //Height of matrix
const static int tile_dim = 16; //Tile size (block dimensions)

//kernel function to perform simple data copy from one array to another
__global__ void copy_kernel(float *in, float *out, int width, int height){
    //compute global thread indices
    int x_index = blockIdx.x * tile_dim + threadIdx.x; //column index
    int y_index = blockIdx.y * tile_dim + threadIdx.y; //row index

    // compute the linear index for 2D matrix
    int index = y_index * width + x_index;

    // copy the element from input array to output array 
    out[index] = in[index];
}


int main(){
    //Host-side vectors for input and output matrices
    std::vector<float> matrix_in;
    std::vector<float> matrix_out;

    //Resize host vectors to match the matrix dimensions
    matrix_in.resize(width * height);
    matrix_out.resize(width * height);

    //Initialize the input matrix with random float values
    for (int i=0; i < width * height; i++){
        matrix_in[i] = (float)rand() / (float)RAND_MAX;
    }

    //Device pointer for input and output matrices
    float *d_in, *d_out;

    //Allocate memory on the device for input and output matrices
    hipMalloc((void **)&d_in, width * height * sizeof(float));
    hipMalloc((void **)&d_out, width * height * sizeof(float));
    
    //copy the input matrix data from host to device
    hipMemcpy(d_in, matrix_in.data(), width * height * sizeof(float), hipMemcpyHostToDevice);


    printf("Setup complete. launching kernel \n");

    //compute the grid  and block dimensions
    int block_x = width / tile_dim;  //Number of blocks in x-dimension
    int block_y = height / tile_dim; //Number of blocks in y-dimension

    //Create CUDA events for timing kernel execution
    hipEvent_t start_kernel_event, end_kernel_event;
    hipEventCreate(&start_kernel_event);
    hipEventCreate(&end_kernel_event);

    //Warm up the GPU by launching the kernel  multiple times (optional, improves timing accuracy)
    printf("Warming up the GPU...\n");
    for (int i = 1; i <= 10; i++){
        copy_kernel<<<dim3(block_x, block_y), dim3(tile_dim, tile_dim)>>>(d_in, d_out, width, height);
    }

    //Start recording the kernel execution time
    hipEventRecord(start_kernel_event, 0);
    
    //launch the kernel multiple times to calculate the average execution time
    for (int i = 1; i <= 10; i++) {
        copy_kernel<<<dim3(block_x, block_y), dim3(tile_dim, tile_dim)>>>(d_in,d_out, width, height);
    }

    //Stop recording the kernel execution time
    hipEventRecord(end_kernel_event, 0);
    hipEventSynchronize(end_kernel_event);

    // Synchronize the device to ensure all kernels have completed execution
    hipDeviceSynchronize();

    //calculate the elapsed time for the kernel
    float time_kernel;
    hipEventElapsedTime(&time_kernel, start_kernel_event, end_kernel_event);

    printf("Kernel execution complete. \n");
    printf("Event timings: \n");
    printf(" %.6f ms - copy \n", time_kernel / 10);

    //calculate memory bandwidth
    double bandwidth = 2.0 * ((double)width * (double)height * sizeof(float)) / (time_kernel * 1024 * 1024 * 1024);
    printf("Bandwidth: %.6f GB/s\n", bandwidth);

    //copy the output matrix from device to host
    hipMemcpy(matrix_out.data(), d_out, width * height * sizeof(float), hipMemcpyDeviceToHost);

    //Free device memory
    hipFree(d_in);
    hipFree(d_out);


    //Destroy CUDA events 
    hipEventDestroy(start_kernel_event);
    hipEventDestroy(end_kernel_event);

    printf("Execution complete. Check results for correctness. \n");

    return 0;



}
